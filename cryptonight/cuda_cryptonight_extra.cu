#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

static uint32_t d_inlen[8];
static uint32_t *d_input[8];
static uint32_t *d_target[8];
static uint32_t *d_resultNonce[8];

#include "cuda_cryptonight_keccak.cu"
#include "cuda_cryptonight_blake.cu"
#include "cuda_cryptonight_groestl.cu"
#include "cuda_cryptonight_jh.cu"
#include "cuda_cryptonight_skein.cu"

__constant__ uint8_t d_sub_byte[16][16];

const uint8_t sub_byte[16][16] = {
    { 0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76 },
    { 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0 },
    { 0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15 },
    { 0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75 },
    { 0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84 },
    { 0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf },
    { 0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8 },
    { 0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2 },
    { 0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73 },
    { 0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb },
    { 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79 },
    { 0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08 },
    { 0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a },
    { 0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e },
    { 0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf },
    { 0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 }
};

__device__ void cryptonight_aes_set_key( uint32_t *key, const uint32_t *data )
{
    int i, j;
    const int key_base = 8;
    uint8_t temp[4];
    const uint32_t aes_gf[] = { 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };
    
    MEMSET4(key, 0, 40);
    MEMCPY4(key, data, 8);

	for( i = key_base; i < 40; i++ ) {

		*(uint32_t *)temp = key[i - 1];
		
		if( i % key_base == 0 ) {

            *(uint32_t *)temp = ROTR32(*(uint32_t *)temp, 8);

			for( j = 0; j < 4; j++ )
                temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];

			*(uint32_t *)temp ^= aes_gf[i / key_base - 1];
		}
		else if( i % key_base == 4 )
            for( j = 0; j < 4; j++ )
                temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];

        key[i] = key[(i - key_base)] ^ *(uint32_t *)temp;
	}
}

__global__ void cryptonight_extra_gpu_prepare(int threads, uint32_t *d_input, int inlen, uint32_t startNonce, struct cryptonight_gpu_ctx *d_ctx)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    
    if (thread < threads)
    {
        struct cryptonight_gpu_ctx ctx;
        uint32_t input[32];

        memcpy(input, d_input, inlen);
        *((uint32_t *)(((char *)input) + 39)) = startNonce + thread;
        
        cn_keccak((uint8_t *)input, inlen, (uint8_t *)ctx.state);
        cryptonight_aes_set_key(ctx.key1, ctx.state);
        cryptonight_aes_set_key(ctx.key2, ctx.state+8);
        XOR_BLOCKS_DST(ctx.state, ctx.state+8, ctx.a);
        XOR_BLOCKS_DST(ctx.state+4, ctx.state+12, ctx.b);

        memcpy(&d_ctx[thread], &ctx, sizeof(struct cryptonight_gpu_ctx));
    }
}

__global__ void cryptonight_extra_gpu_final(int threads, uint32_t startNonce, uint32_t *d_target, uint32_t *resNonce, struct cryptonight_gpu_ctx *d_ctx)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
   
    if (thread < threads)
    {
        MY_ALIGN(8)uint32_t state[50];
        int i;
        uint32_t nonce = startNonce + thread;
        struct cryptonight_gpu_ctx *ctx = &d_ctx[thread];
        uint32_t hash[8];

        MEMCPY8(state, &ctx->state, 25);
        cn_keccakf((uint64_t *)state);

        switch( ((uint8_t *)state)[0] & 0x03 ) {
            case 0:
                cn_blake((const uint8_t *)state, 200, (uint8_t *)hash);
                break;
            case 1:
                cn_groestl((const BitSequence *)state, 200, (BitSequence *)hash);
                break;
            case 2:
                cn_jh((const BitSequence *)state, 200, (BitSequence *)hash);
                break;
            case 3:
                cn_skein((const BitSequence *)state, 200, (BitSequence *)hash);
                break;
            default:
                break;
        }

        int position = -1;
        bool rc = true;

#if 0
#pragma unroll 8
#endif
        for (i = 7; i >= 0; i--) {
            if (hash[i] > d_target[i]) {
                if(position < i) {
                    position = i;
                    rc = false;
                }
             }
             if (hash[i] < d_target[i]) {
                if(position < i) {
                    position = i;
                    rc = true;
                }
             }
        }

        if(rc == true)
            if(resNonce[0] > nonce)
                resNonce[0] = nonce;
    }
}

__host__ void cryptonight_extra_cpu_setData(int thr_id, const void *data, const void *pTargetIn)
{
    hipMemcpy(d_input[thr_id], data, d_inlen[thr_id], hipMemcpyHostToDevice);
    hipMemcpy(d_target[thr_id], pTargetIn, 8*sizeof(uint32_t), hipMemcpyHostToDevice);
}

__host__ void cryptonight_extra_cpu_init(int thr_id, int inlen)
{
    d_inlen[thr_id] = inlen;
    hipMalloc(&d_input[thr_id], inlen); 
    hipMalloc(&d_target[thr_id], 8*sizeof(uint32_t)); 
    hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
    hipMemcpyToSymbol(HIP_SYMBOL(keccakf_rndc), h_keccakf_rndc, sizeof(h_keccakf_rndc), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sub_byte), sub_byte, sizeof(sub_byte), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_blake_sigma), h_blake_sigma, sizeof(h_blake_sigma), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_blake_cst), h_blake_cst, sizeof(h_blake_cst), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_groestl_T), h_groestl_T, sizeof(h_groestl_T), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_JH256_H0), h_JH256_H0, sizeof(h_JH256_H0), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_E8_rc), h_E8_rc, sizeof(h_E8_rc), 0, hipMemcpyHostToDevice);
}

__host__ void cryptonight_extra_cpu_prepare(int thr_id, int threads, uint32_t startNonce, struct cryptonight_gpu_ctx *d_ctx)
{
    int threadsperblock = 128;
    size_t shared_size = 0;

    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    cryptonight_extra_gpu_prepare<<<grid, block, shared_size>>>(threads, d_input[thr_id], d_inlen[thr_id], startNonce, d_ctx);
    hipDeviceSynchronize();
}

__host__ void cryptonight_extra_cpu_final(int thr_id, int threads, uint32_t startNonce, uint32_t *resnonce, struct cryptonight_gpu_ctx *d_ctx)
{
    int threadsperblock = 128;
    size_t shared_size = 0;

    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    
    cryptonight_extra_gpu_final<<<grid, block, shared_size>>>(threads, startNonce, d_target[thr_id], d_resultNonce[thr_id], d_ctx);
    hipDeviceSynchronize();

    hipMemcpy(resnonce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}

