#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cryptonight.h"

#ifndef _WIN32
#include <unistd.h>
#endif

typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

static uint32_t *d_input[MAX_GPU];
static uint32_t *d_target[MAX_GPU];
static uint32_t *d_resultNonce[MAX_GPU];

#include "cuda_cryptonight_keccak.cu"
#include "cuda_cryptonight_blake.cu"
#include "cuda_cryptonight_groestl.cu"
#include "cuda_cryptonight_jh.cu"
#include "cuda_cryptonight_skein.cu"

__constant__ uint8_t d_sub_byte[16][16] =
{
	{0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76},
	{0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0},
	{0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15},
	{0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75},
	{0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84},
	{0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf},
	{0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8},
	{0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2},
	{0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73},
	{0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb},
	{0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79},
	{0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08},
	{0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a},
	{0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e},
	{0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf},
	{0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16}
};

__device__ __forceinline__ void cryptonight_aes_set_key(uint32_t * __restrict__ key, const uint32_t * __restrict__ data)
{
	int i, j;
	uint8_t temp[4];
	const uint32_t aes_gf[10] =
	{
		0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36
	};

	MEMCPY4(key, data, 8);
#pragma unroll
	for(i = 8; i < 40; i++)
	{
		*(uint32_t *)temp = key[i - 1];
		if(i % 8 == 0)
		{
			*(uint32_t *)temp = ROTR32(*(uint32_t *)temp, 8);
			for(j = 0; j < 4; j++)
				temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];
			*(uint32_t *)temp ^= aes_gf[i / 8 - 1];
		}
		else
			if(i % 8 == 4)
#pragma unroll
				for(j = 0; j < 4; j++)
					temp[j] = d_sub_byte[(temp[j] >> 4) & 0x0f][temp[j] & 0x0f];
		key[i] = key[(i - 8)] ^ *(uint32_t *)temp;
	}
}

__global__ void cryptonight_extra_gpu_prepare(int threads, const uint32_t * __restrict__ d_input, uint32_t startNonce, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b, uint32_t * __restrict__ d_ctx_key1, uint32_t * __restrict__ d_ctx_key2, int variant, uint32_t * d_ctx_tweak1_2)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if(thread < threads)
	{
		uint64_t ctx_state[25];
		uint32_t ctx_a[4];
		uint32_t ctx_b[4];
		uint32_t ctx_key1[40] = {0};
		uint32_t ctx_key2[40] = {0};
		uint32_t input[19];
		uint32_t tweak1_2[2];

		MEMCPY4(input, d_input, 19);

		uint32_t nonce = startNonce + thread;
		*(((uint8_t *)input) + 39) = nonce & 0xff;
		*(((uint8_t *)input) + 40) = (nonce >> 8) & 0xff;
		*(((uint8_t *)input) + 41) = (nonce >> 16) & 0xff;
		*(((uint8_t *)input) + 42) = (nonce >> 24) & 0xff;

		cn_keccak(input, ctx_state);
		cryptonight_aes_set_key(ctx_key1, (uint32_t*)ctx_state);
		cryptonight_aes_set_key(ctx_key2, (uint32_t*)(ctx_state + 4));
		XOR_BLOCKS_DST(ctx_state, ctx_state + 4, ctx_a);
		XOR_BLOCKS_DST(ctx_state + 2, ctx_state + 6, ctx_b);

                if (variant > 0)
		{
			tweak1_2[0] = (input[8] >> 24) | (input[9] << 8);
			tweak1_2[0] ^= (ctx_state[24] & 0xffffffff);
			tweak1_2[1] = (input[9] >> 24) | (input[10] << 8);
			tweak1_2[1] ^= (ctx_state[24] >> 32);
			MEMCPY4(d_ctx_tweak1_2 + thread * 2, tweak1_2, 2);
		}

		MEMCPY4(d_ctx_state + thread * 50, ctx_state, 50);
		MEMCPY4(d_ctx_a + thread * 4, ctx_a, 4);
		MEMCPY4(d_ctx_b + thread * 4, ctx_b, 4);
		MEMCPY4(d_ctx_key1 + thread * 40, ctx_key1, 40);
		MEMCPY4(d_ctx_key2 + thread * 40, ctx_key2, 40);
	}
}

__global__ void cryptonight_extra_gpu_final(int threads, uint32_t startNonce, const uint32_t * __restrict__ d_target, uint32_t * __restrict__ resNonce, uint32_t * __restrict__ d_ctx_state)
{
	const int thread = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(thread < threads)
	{
		int i;
		const uint32_t nonce = startNonce + thread;
		const uint32_t * __restrict__ ctx_state = d_ctx_state + thread * 50;
		uint32_t hash[8];
		uint32_t state[50];

#pragma unroll
		for(i = 0; i < 50; i++)
			state[i] = ctx_state[i];

		cn_keccakf2((uint64_t *)state);

		int branch = ((uint8_t *)state)[0] & 0x03;
		if(branch == 0)
			cn_blake((const uint8_t *)state, 200, (uint8_t *)hash);
		if(branch == 1)
			cn_groestl((const BitSequence *)state, 200, (BitSequence *)hash);
		if(branch == 2)
			cn_jh((const BitSequence *)state, 200, (BitSequence *)hash);
		if(branch == 3)
			cn_skein((const BitSequence *)state, 200, (BitSequence *)hash);

		int position = -1;
		bool rc = true;

#pragma unroll 8
		for(i = 7; i >= 0; i--)
		{
			if(hash[i] > d_target[i])
			{
				if(position < i)
				{
					position = i;
					rc = false;
				}
			}
			if(hash[i] < d_target[i])
			{
				if(position < i)
				{
					position = i;
					rc = true;
				}
			}
		}

		if(rc == true)
		{
			uint32_t tmp = atomicExch(resNonce, nonce);
			if(tmp != 0xffffffff)
				resNonce[1] = tmp;
		}
	}
}

__host__ void cryptonight_extra_cpu_setData(int thr_id, const void *data, const void *pTargetIn)
{
	hipMemcpy(d_input[thr_id], data, 19 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(d_target[thr_id], pTargetIn, 8 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemset(d_resultNonce[thr_id], 0xFF, 2 * sizeof(uint32_t));
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
}

__host__ void cryptonight_extra_cpu_init(int thr_id)
{
	hipMalloc(&d_input[thr_id], 19 * sizeof(uint32_t));
	hipMalloc(&d_target[thr_id], 8 * sizeof(uint32_t));
	hipMalloc(&d_resultNonce[thr_id], 2*sizeof(uint32_t));
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
}

__host__ void cryptonight_extra_cpu_prepare(int thr_id, int threads, uint32_t startNonce, uint32_t *d_ctx_state, uint32_t *d_ctx_a, uint32_t *d_ctx_b, uint32_t *d_ctx_key1, uint32_t *d_ctx_key2, int variant, uint32_t *d_ctx_tweak1_2)
{
	int threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cryptonight_extra_gpu_prepare << <grid, block >> >(threads, d_input[thr_id], startNonce, d_ctx_state, d_ctx_a, d_ctx_b, d_ctx_key1, d_ctx_key2, variant, d_ctx_tweak1_2);
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
}

__host__ void cryptonight_extra_cpu_final(int thr_id, int threads, uint32_t startNonce, uint32_t *resnonce, uint32_t *d_ctx_state)
{
	int threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
	cryptonight_extra_gpu_final << <grid, block >> >(threads, startNonce, d_target[thr_id], d_resultNonce[thr_id], d_ctx_state);
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
	hipMemcpy(resnonce, d_resultNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	exit_if_cudaerror(thr_id, __FILE__, __LINE__);
}
