#include "hip/hip_runtime.h"
#include <ctype.h>
#include <unistd.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

extern "C"
{
#include "cpuminer-config.h"
#include "miner.h"
#include "cryptonight.h"
}

extern char *device_name[8];
extern int device_arch[8][2];
extern int device_mpcount[8];
extern int device_map[8];
extern int device_config[8][2];

// Zahl der CUDA Devices im System bestimmen
extern "C" int cuda_num_devices()
{
    int version;
    hipError_t err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query CUDA driver version! Is an nVidia driver installed?");
        exit(1);
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < 5 || (maj == 5 && min < 5))
    {
        applog(LOG_ERR, "Driver does not support CUDA %d.%d API! Update your nVidia driver!", 5, 5);
        exit(1);
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }
    return GPU_N;
}

extern "C" void cuda_deviceinfo()
{
    hipError_t err;
    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "Unable to query number of CUDA devices! Is an nVidia driver installed?");
        exit(1);
    }

    for (int i=0; i < GPU_N; i++)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, device_map[i]);

        device_name[i] = strdup(props.name);
        device_mpcount[i] = props.multiProcessorCount;
        device_arch[i][0] = props.major;
        device_arch[i][1] = props.minor;
    }
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
    int hlen = strlen(haystack);
    int nlen = strlen(needle);
    for (int i=0; i < hlen; ++i)
    {
        if (haystack[i] == ' ') continue;
        int j=0, x = 0;
        while(j < nlen)
        {
            if (haystack[i+x] == ' ') {++x; continue;}
            if (needle[j] == ' ') {++j; continue;}
            if (needle[j] == '#') return ++match == needle[j+1]-'0';
            if (tolower(haystack[i+x]) != tolower(needle[j])) break;
            ++j; ++x;
        }
        if (j == nlen) return true;
    }
    return false;
}

// CUDA Ger�t nach Namen finden (gibt Ger�te-Index zur�ck oder -1)
extern "C" int cuda_finddevice(char *name)
{
    int num = cuda_num_devices();
    int match = 0;
    for (int i=0; i < num; ++i)
    {
        hipDeviceProp_t props;
        if (hipGetDeviceProperties(&props, i) == hipSuccess)
            if (substringsearch(props.name, name, match)) return i;
    }
    return -1;
}

static uint32_t *d_long_state[8];
static struct cryptonight_gpu_ctx *d_ctx[8];

extern bool opt_benchmark;

extern void cryptonight_core_cpu_init(int thr_id, int threads);
extern void cryptonight_core_cpu_hash(int thr_id, int blocks, int threads, uint32_t *d_long_state, struct cryptonight_gpu_ctx *d_ctx);

extern void cryptonight_extra_cpu_setData(int thr_id, const void *data, const void *pTargetIn);
extern void cryptonight_extra_cpu_init(int thr_id, int dlen);
extern void cryptonight_extra_cpu_prepare(int thr_id, int threads, uint32_t startNonce, struct cryptonight_gpu_ctx *d_ctx);
extern void cryptonight_extra_cpu_final(int thr_id, int threads, uint32_t startNonce, uint32_t *nonce, struct cryptonight_gpu_ctx *d_ctx);

extern "C" void cryptonight_hash(void* output, const void* input, size_t len);

extern "C" int scanhash_cryptonight(int thr_id, uint32_t *pdata, int dlen,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
    uint32_t *nonceptr = (uint32_t*)(((char*)pdata) + 39);
    const uint32_t first_nonce = *nonceptr;
    uint32_t nonce = *nonceptr;
    int cn_blocks = device_config[thr_id][0], cn_threads = device_config[thr_id][1];

	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x0000ff;
        pdata[17] = 0;
    }
	const uint32_t Htarg = ptarget[7];
	const int throughput = cn_threads * cn_blocks;
    const size_t alloc = MEMORY * throughput;

    static bool init[8] = { false, false, false, false, false, false, false, false };
	if (!init[thr_id])
	{
        hipSetDevice(device_map[thr_id]);
        hipDeviceReset();
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if( hipMalloc(&d_long_state[thr_id], alloc) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for long state", thr_id);
            exit(1);
        }
		if( hipMalloc(&d_ctx[thr_id], sizeof(struct cryptonight_gpu_ctx) * throughput) != hipSuccess ) {
            applog(LOG_ERR, "GPU #%d: FATAL: failed to allocate device memory for hash context", thr_id);
            exit(1);
        }
		cryptonight_core_cpu_init(thr_id, throughput);
        cryptonight_extra_cpu_init(thr_id, dlen);
		init[thr_id] = true;
	}

    cryptonight_extra_cpu_setData(thr_id, (const void *)pdata, (const void *)ptarget);

	do {
        uint32_t foundNonce = 0xFFFFFFFF;

        cryptonight_extra_cpu_prepare(thr_id, throughput, nonce, d_ctx[thr_id]);
        cryptonight_core_cpu_hash(thr_id, cn_blocks, cn_threads, d_long_state[thr_id], d_ctx[thr_id]);
        cryptonight_extra_cpu_final(thr_id, throughput, nonce, &foundNonce, d_ctx[thr_id]);

        if (foundNonce < 0xffffffff)
		{
			uint32_t vhash64[8];
            uint32_t tempdata[32];
            uint32_t *tempnonceptr = (uint32_t*)(((char*)tempdata) + 39);
            memcpy(tempdata, pdata, dlen);
			*tempnonceptr = foundNonce;
			cryptonight_hash(vhash64, tempdata, dlen);

            if( (vhash64[7] <= Htarg) && fulltest(vhash64, ptarget) ) {
                
			    *nonceptr = foundNonce;
                *hashes_done = foundNonce - first_nonce + 1;
                return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		
            foundNonce = 0xffffffff;
        }

		nonce += throughput;
	} while (nonce < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = nonce - first_nonce + 1;
	return 0;
}
